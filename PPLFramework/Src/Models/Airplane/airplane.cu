#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <random>
#include <time.h>

#include "../../Inference/Smc/smc.cuh"
#include "../../Inference/Smc/smcImpl.cuh"
#include "../../Utils/distributions.cuh"
#include "../../Utils/misc.cuh"
#include "airplane.cuh"
#include "airplaneUtils.cuh"

// nvcc -arch=sm_75 -rdc=true Src/Models/Airplane/*.cu -o smc.exe -lcudadevrt -std=c++11 -O3 -D GPU

using namespace std;

#define NUM_BBLOCKS 2
INIT_GLOBAL(progState_t, NUM_BBLOCKS)

floating_t planeX[TIME_STEPS];

BBLOCK_DATA(planeObs, floating_t, TIME_STEPS)
BBLOCK_DATA(mapApprox, floating_t, MAP_SIZE)


void initAirplane() {

    initMap(mapApprox);

    initObservations(planeX, planeObs, mapApprox);

    // Copy data to device pointers, so that they can be accessed from kernels
    COPY_DATA_GPU(planeObs, floating_t, TIME_STEPS)
    COPY_DATA_GPU(mapApprox, floating_t, MAP_SIZE)
}

BBLOCK(propagateAndWeight, progState_t, {

    // Propagate
    PSTATE.x += BBLOCK_CALL(sampleNormal, VELOCITY, TRANSITION_STD);

    // Weight
    WEIGHT(logNormalPDFObs(DATA_POINTER(planeObs)[PSTATE.t], mapLookupApprox(DATA_POINTER(mapApprox), PSTATE.x)));
    PSTATE.t++;

    if(PSTATE.t >= TIME_STEPS - 1)
        PC = 2;

    // RESAMPLE = true;
})

BBLOCK(particleInit, progState_t, {

    PSTATE.x = BBLOCK_CALL(sampleUniform, 0, MAP_SIZE);
    PSTATE.t = 0;

    PC = 1;
    // RESAMPLE = false;
    BBLOCK_CALL(propagateAndWeight);
})

CALLBACK_HOST(callback, progState_t, {
    // Checks how many particles are close to actual airplane to check for correctness
    int numParticlesClose = 0;
    floating_t minX = 999999;
    floating_t maxX = -1;
    for (int i = 0; i < NUM_PARTICLES; i++) {
        floating_t particleX = PSTATE.x;
        if(abs(particleX - planeX[TIME_STEPS-1]) < 10)
            numParticlesClose++;
        minX = min(minX, particleX);
        maxX = max(maxX, particleX);
    }

    cout << "Num particles close to target: " << 100 * static_cast<floating_t>(numParticlesClose) / NUM_PARTICLES << "%, MinX: " << minX << ", MaxX: " << maxX << endl;
}, void* arg = NULL)

int main(int argc, char** argv) {

    initAirplane();

    // SMCSTART(progState_t, NUM_BBLOCKS)

    INITBBLOCK(particleInit, progState_t)
    INITBBLOCK(propagateAndWeight, progState_t)

    SMC(progState_t, callback)
}
